#include <iostream>
#include <cmath>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

__global__
void addKernel(int n, float *x, float *y)
{
    // This is a so called grid-stride-loop, which ensures
    // that add addressing within warps is unit-stride, and thus
    // achieves maximum memory coalescing.
    // For more information: https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    // and https://developer.nvidia.com/blog/how-access-global-memory-efficiently-cuda-c-kernels/
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

int main(int argc, char** argv)
{
    int N = 1 << 20; //1M elements

    float *x, *y;

    // Allocate memory, uses unified memory
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // Initialize the memory on the host device
    for( int i = 0; i < N; ++i)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run the kernel on the elements
    int blockSize = 256; // For efficiency, this needs to be a multiple of 32
    int numBlocks = (N + blockSize -1) / blockSize;
    
    std::cout << "Number of elements: " << N << std::endl;
    std::cout << "Block size: " << blockSize << std::endl;
    std::cout << "Number of blocks: " << numBlocks << std::endl;
    
    addKernel<<<numBlocks, blockSize>>>(N, x, y);

    // Wait for all the kernels to finnish execution
    hipDeviceSynchronize();

    //Check for errors (all values should be 3.0f)
    float maxError = 0.0f;

    for( int i = 0; i < N; ++i )
    {
        maxError = std::max(maxError, std::abs(y[i] - 3.0f));
    }

    hipFree(x);
    hipFree(y);

    std::cout << "Max error: " << maxError << std::endl;

    hipDeviceReset();
    return 0;
}
