#include <iostream>
#include <cmath>

#include <stdio.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

__global__
void print_vector_values(float* vector)
{
    // This is a so called monolithic kernel, which assumes
    // that the grid of threads processes the array in a single pass.

     // Row stride
     int row_stride = blockDim.x * gridDim.x;

    // Block offset is the offset to the beginning of the block in question
    int block_offset = blockDim.x * blockIdx.x  + row_stride * blockDim.y * blockIdx.y;

    //Global index
    int gidx =  block_offset + row_stride * threadIdx.y + threadIdx.x;

    printf("blockIdx: (%d, %d), threadIdx: (%d, %d), value: %f\n",
           blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, vector[gidx]);
}

int main(int argc, char** argv)
{
    int nx, ny;
    nx = 8; // Total number of threads in x-direction
    ny = 4; // Total number of threads in y-direction

    dim3 block(4, 2); // Number of threads per block
    dim3 grid(nx / block.x, ny / block.y);

    // Create a vector in the host. This vector represents
    // the following 2D data:
    // 11, 12, 13, 14, 15, 16, 17, 18
    // 21, 22, 23, 24, 25, 26, 27, 28
    // 31, 32, 33, 34, 35, 36, 37, 38
    // 41, 42, 43, 44, 45, 46, 47, 48
    using vector_t = float;
    vector_t vector[] = {
        11, 12, 13, 14, 15, 16, 17, 18, // 8 elements
        21, 22, 23, 24, 25, 26, 27, 28, // 8 elements
        31, 32, 33, 34, 35, 36, 37, 38, // 8 elements
        41, 42, 43, 44, 45, 46, 47, 48 }; // 8 elements

    // Reserve memory in the GPU
    vector_t *vector_gpu;
    hipMalloc(reinterpret_cast<void**>(&vector_gpu), sizeof(vector_t)*32);

    // Copy data from host to the GPU
    hipMemcpy(vector_gpu, &vector, sizeof(vector_t)*32, hipMemcpyHostToDevice);

    // Print the values of the input vector
    std::cout << "Accessing the vector in the device:" << std::endl;
    print_vector_values<<<grid, block>>>(vector_gpu);

    // Wait for all the kernels to finnish execution
    hipDeviceSynchronize();

    // Free memory
    hipFree(vector_gpu);

    hipDeviceReset();
    return 0;
}
