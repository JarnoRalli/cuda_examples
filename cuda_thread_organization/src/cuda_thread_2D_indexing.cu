#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#include "tools/timing.hpp"
#include "tools/error_handling.hpp"

// Vector element type
using vector_t = float;

__global__
void print_vector_values(vector_t* vector)
{
    // This is a so called monolithic kernel, which assumes
    // that the grid of threads processes the array in a single pass.

     // Row stride
     int row_stride = blockDim.x * gridDim.x;

    // Block offset is the offset to the beginning of the block in question
    int block_offset = blockDim.x * blockIdx.x  + row_stride * blockDim.y * blockIdx.y;

    //Global index
    int gidx =  block_offset + row_stride * threadIdx.y + threadIdx.x;

    printf("blockIdx: (%d, %d), threadIdx: (%d, %d), value: %f\n",
           blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, vector[gidx]);
}

int main(int argc, char** argv)
{
    int nx, ny;
    nx = 8; // Total number of threads in x-direction
    ny = 4; // Total number of threads in y-direction

    dim3 block(4, 2); // Number of threads per block
    dim3 grid(nx / block.x, ny / block.y);

    // Create a vector in the host. This vector represents
    // the following 2D data:
    // 11, 12, 13, 14, 15, 16, 17, 18
    // 21, 22, 23, 24, 25, 26, 27, 28
    // 31, 32, 33, 34, 35, 36, 37, 38
    // 41, 42, 43, 44, 45, 46, 47, 48
    vector_t vector[] = {
        11, 12, 13, 14, 15, 16, 17, 18, // 8 elements
        21, 22, 23, 24, 25, 26, 27, 28, // 8 elements
        31, 32, 33, 34, 35, 36, 37, 38, // 8 elements
        41, 42, 43, 44, 45, 46, 47, 48 }; // 8 elements

    // Reserve memory in the GPU
    vector_t *vector_gpu;
    gpuErrchk(hipMalloc(reinterpret_cast<void**>(&vector_gpu), sizeof(vector_t)*32));

    // For instrumentation
    auto start_time = std::chrono::steady_clock::now();

    // Copy data from host to the GPU
    gpuErrchk(hipMemcpy(vector_gpu, &vector, sizeof(vector_t)*32, hipMemcpyHostToDevice));
    auto host2device_cpy_duration = since(start_time);

    // Print the values of the input vector
    std::cout << "Accessing the vector in the device:" << std::endl;
    start_time = std::chrono::steady_clock::now();
    print_vector_values<<<grid, block>>>(vector_gpu);

    // Wait for all the kernels to finnish execution
    gpuErrchk(hipDeviceSynchronize());
    auto execution_duration = since(start_time);

    // Print memory transfer and kernel execution times
    std::cout << "Host to device memcpy time (ns) : " << host2device_cpy_duration.count() << std::endl;
    std::cout << "Kernel execution time (ns) : " << execution_duration.count() << std::endl;

    // Free memory
    gpuErrchk(hipFree(vector_gpu));

    gpuErrchk(hipDeviceReset());
    return 0;
}
