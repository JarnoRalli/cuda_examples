#include <iostream>
#include <cmath>

#include <stdio.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

__global__
void print_vector_values(float* vector)
{
    // This is a so called monolithic kernel, which assumes
    // that the grid of threads processes the array in a single pass.
    
    // Thread index
    int tidx = threadIdx.x;

    // Block index
    int bidx = blockIdx.x;

    //Global index
    int gidx = blockDim.x * bidx + tidx;

    printf("blockIdx.x: %d, threadIdx.x: %d, vector[%d] = %f\n", bidx, tidx, gidx, vector[gidx]);
}

int main(int argc, char** argv)
{
    int nx;
    nx = 8; // Total number of threads in x-direction

    dim3 block(4); // Number of threads per block
    dim3 grid(nx / block.x);

    // Create a vector in the host
    using vector_t = float;
    vector_t *vector;
    vector = new vector_t [nx];

    std::cout << "Vector in the host: ";
    for(int i = 0; i < nx; ++i )
    {
        vector[i] = static_cast<float>(i) + 0.1f;
        if( i != nx -1 )
        {
            std::cout << vector[i] << ", ";
        }else{
            std::cout << vector[i] << std::endl;
        }
    }

    // Reserve memory in the GPU
    vector_t *vector_gpu;
    hipMalloc(reinterpret_cast<void**>(&vector_gpu), sizeof(vector_t)*nx);

    // Copy data from host to the GPU
    hipMemcpy(vector_gpu, vector, sizeof(vector_t)*nx, hipMemcpyHostToDevice);

    // Print the values of the input vector
    std::cout << "Accessing the vector in the device:" << std::endl;
    print_vector_values<<<grid, block>>>(vector_gpu);

    // Wait for all the kernels to finnish execution
    hipDeviceSynchronize();

    // Free memory
    hipFree(vector_gpu);
    delete vector;

    hipDeviceReset();
    return 0;
}
